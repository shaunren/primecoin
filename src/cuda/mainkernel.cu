#include "hip/hip_runtime.h"
#include <stdio.h>

#include "mpz.h"    // multiple precision cuda code
#include "cuda_string.h"


struct cudaCandidateTransfer {
    char strChainOrigin[256];
    unsigned int blocknBits;
    unsigned int nChainLengthCunningham1;
    unsigned int nChainLengthCunningham2;
    unsigned int nChainLengthBiTwin;
};

__device__ mpz_t mpzOne;
__device__ mpz_t mpzTwo;
__device__ mpz_t mpzEight;

//__device__ mpz_t mpzTemp;

#define mpz_clear mpz_destroy
#define mpz_cmp mpz_compare
#define mpz_mul mpz_mult
#define mpz_powm mpz_powmod

//copied constants from prime.h

static const unsigned int nFractionalBits = 24;
static const unsigned int TARGET_FRACTIONAL_MASK = (1u<<nFractionalBits) - 1;
static const unsigned int TARGET_LENGTH_MASK = ~TARGET_FRACTIONAL_MASK;
//static const uint64 nFractionalDifficultyMax = (1llu << (nFractionalBits + 32));
//static const uint64 nFractionalDifficultyMin = (1llu << 32);
//static const uint64 nFractionalDifficultyThreshold = (1llu << (8 + 32));
//static const unsigned int nWorkTransitionRatio = 32;

//end copy

//mpz_div(mpz_t *q, mpz_t *r, mpz_t *n, mpz_t *d)


//extra mpz_functions (quick and dirty...)
__device__ inline void mpz_tdiv_q(mpz_t *ROP, mpz_t *OP1, mpz_t *OP2)
{
    mpz_t mpzTemp;
    mpz_init(&mpzTemp);
    mpz_div(ROP,&mpzTemp,OP1,OP2);
    mpz_destroy(&mpzTemp);
}

__device__ inline void mpz_tdiv_r(mpz_t *ROP, mpz_t *OP1, mpz_t *OP2)
{
    mpz_t mpzTemp;
    mpz_init(&mpzTemp);
    mpz_div(&mpzTemp,ROP,OP1,OP2);
    mpz_destroy(&mpzTemp);
}

__device__ inline unsigned int mpz_get_ui(mpz_t *OP)
{
    return OP->capacity==0 ? 0 : OP->digits[0];
}

//Set product to multiplicator times 2 raised to exponent_of_2. This operation can also be defined as a left shift, exponent_of_2 steps.
__device__ inline void mpz_mul_2exp (mpz_t *product, mpz_t *multiplicator, unsigned long int exponent_of_2)
{
    mpz_t mpzTemp;
    mpz_init(&mpzTemp);
    mpz_set(&mpzTemp,&mpzTwo);
    unsigned int limit = exponent_of_2;
    //well this is ugly
    for(unsigned int i=0; i < limit; i++)
    	mpz_bit_lshift(&mpzTemp);

    mpz_mul(product,multiplicator,&mpzTemp);
    mpz_destroy(&mpzTemp);
}


//end extra mpz

__device__ bool devTargetSetLength(unsigned int nLength, unsigned int& nBits)
{
    if (nLength >= 0xff)
    {
        printf("[CUDA] error TargetSetLength() : invalid length=%u", nLength);
	return false;
    }
    nBits &= TARGET_FRACTIONAL_MASK;
    nBits |= (nLength << nFractionalBits);
    return true;
}

__device__ unsigned int devTargetGetLength(unsigned int nBits)
{
    return ((nBits & TARGET_LENGTH_MASK) >> nFractionalBits);
}

__device__ unsigned int devTargetFromInt(unsigned int nLength)
{
    return (nLength << nFractionalBits);
}

__device__ void devTargetIncrementLength(unsigned int& nBits)
{
    nBits += (1 << nFractionalBits);
}

// Check Fermat probable primality test (2-PRP): 2 ** (n-1) = 1 (mod n)
// true: n is probable prime
// false: n is composite; set fractional length in the nLength output
__device__ bool devFermatProbablePrimalityTest(mpz_t &mpzN, unsigned int& nLength)
{
    // Faster GMP version
    
    //mpz_t mpzN;
    mpz_t mpzE;
    mpz_t mpzR;
    
    //mpz_init_set(mpzN, n.get_mpz_t());
    mpz_init(&mpzE);
    mpz_sub(&mpzE, &mpzN, &mpzOne);
    mpz_init(&mpzR);
    mpz_powm(&mpzR, &mpzTwo, &mpzE, &mpzN);
    if (mpz_cmp(&mpzR, &mpzOne) == 0)
    {
        mpz_clear(&mpzN);
        mpz_clear(&mpzE);
        mpz_clear(&mpzR);
        return true;
    }
    // Failed Fermat test, calculate fractional length
    mpz_sub(&mpzE, &mpzN, &mpzR);
    mpz_mul_2exp(&mpzR, &mpzE, nFractionalBits);
    mpz_tdiv_q(&mpzE, &mpzR, &mpzN);

    //todo implement mpz_get_ui
    unsigned int nFractionalLength = mpz_get_ui(&mpzE);
    mpz_clear(&mpzN);
    mpz_clear(&mpzE);
    mpz_clear(&mpzR);

    if (nFractionalLength >= (1 << nFractionalBits))
    {
	printf("[CUDA] Error FermatProbablePrimalityTest() : fractional assert - %d", nFractionalLength);
        return false;
    }

    nLength = (nLength & TARGET_LENGTH_MASK) | nFractionalLength;
    return false;
}

// Test probable primality of n = 2p +/- 1 based on Euler, Lagrange and Lifchitz
// fSophieGermain:
//   true:  n = 2p+1, p prime, aka Cunningham Chain of first kind
//   false: n = 2p-1, p prime, aka Cunningham Chain of second kind
// Return values
//   true: n is probable prime
//   false: n is composite; set fractional length in the nLength output
__device__ bool devEulerLagrangeLifchitzPrimalityTest(mpz_t &mpzN, bool fSophieGermain, unsigned int& nLength)
{
    // Faster GMP version
    //mpz_t mpzN;
    mpz_t mpzE;
    mpz_t mpzR;
    mpz_t temp;

    mpz_init(&temp);    

    mpz_init(&mpzE);
    mpz_sub(&mpzE, &mpzN, &mpzOne);
 
   //e = (n - 1) >> 1;
    //from hp4: mpz_tdiv_q_2exp(&mpzE, &mpzE, 1);
    mpz_tdiv_q(&temp,&mpzE,&mpzTwo);
    mpz_set(&mpzE,&temp);

    mpz_destroy(&temp);

    mpz_init(&mpzR);
    mpz_powm(&mpzR, &mpzTwo, &mpzE, &mpzN);
   
    //nMod8 = n % 8; 
    mpz_t mpzNMod8;
    mpz_init(&mpzNMod8);
    mpz_tdiv_r(&mpzNMod8,&mpzN, &mpzEight);
    unsigned int nMod8 = mpz_get_ui(&mpzNMod8);    
    mpz_destroy(&mpzNMod8);

    bool fPassedTest = false;
    if (fSophieGermain && (nMod8 == 7)) // Euler & Lagrange
        fPassedTest = !mpz_cmp(&mpzR, &mpzOne);
    else if (fSophieGermain && (nMod8 == 3)) // Lifchitz
    {
        mpz_t mpzRplusOne;
        mpz_init(&mpzRplusOne);
        mpz_add(&mpzRplusOne, &mpzR, &mpzOne);
        fPassedTest = !mpz_cmp(&mpzRplusOne, &mpzN);
        mpz_clear(&mpzRplusOne);
    }
    else if ((!fSophieGermain) && (nMod8 == 5)) // Lifchitz
    {
        mpz_t mpzRplusOne;
        mpz_init(&mpzRplusOne);
        mpz_add(&mpzRplusOne, &mpzR, &mpzOne);
        fPassedTest = !mpz_cmp(&mpzRplusOne, &mpzN);
        mpz_clear(&mpzRplusOne);
    }
    else if ((!fSophieGermain) && (nMod8 == 1)) // LifChitz
    {
        fPassedTest = !mpz_cmp(&mpzR, &mpzOne);
    }
    else
    {
        mpz_clear(&mpzN);
        mpz_clear(&mpzE);
        mpz_clear(&mpzR);
        printf("[CUDA] Error in EulerLagrangeLifchitzPrimalityTest() : invalid n %% 8 = %d, %s", nMod8, (fSophieGermain? "first kind" : "second kind"));
        return false;
    }
    
    if (fPassedTest)
    {
        mpz_clear(&mpzN);
        mpz_clear(&mpzE);
        mpz_clear(&mpzR);
        return true;
    }
    
    // Failed test, calculate fractional length
    //TODO: RCOPY
    mpz_mul(&mpzE, &mpzR, &mpzR);
    mpz_tdiv_r(&mpzR, &mpzE, &mpzN); // derive Fermat test remainder

    mpz_sub(&mpzE, &mpzN, &mpzR);
    mpz_mul_2exp(&mpzR, &mpzE, nFractionalBits);
    mpz_tdiv_q(&mpzE, &mpzR, &mpzN);

    //Todo: implement mpz_get_ui
    unsigned int nFractionalLength = mpz_get_ui(&mpzE);
    mpz_clear(&mpzN);
    mpz_clear(&mpzE);
    mpz_clear(&mpzR);
    
    if (nFractionalLength >= (1 << nFractionalBits))
    {
        printf("[CUDA] error EulerLagrangeLifchitzPrimalityTest() : fractional assert - %d", nFractionalLength);
        return false;
    }
    nLength = (nLength & TARGET_LENGTH_MASK) | nFractionalLength;
    return false;
}



// Test Probable Cunningham Chain for: n
// fSophieGermain:
//   true - Test for Cunningham Chain of first kind (n, 2n+1, 4n+3, ...)
//   false - Test for Cunningham Chain of second kind (n, 2n-1, 4n-3, ...)
// Return value:
//   true - Probable Cunningham Chain found (length at least 2)
//   false - Not Cunningham Chain
__device__ bool devProbableCunninghamChainTest(mpz_t &n, bool fSophieGermain, bool fFermatTest, unsigned int& nProbableChainLength)
{
    nProbableChainLength = 0;
    //mpz_class N = n;

    mpz_t N;
    mpz_init(&N);

    mpz_t N_copy;
    mpz_init(&N_copy);

    mpz_set(&N,&n);    

    // Fermat test for n first
    if (!devFermatProbablePrimalityTest(N, nProbableChainLength))
        return false;

    // Euler-Lagrange-Lifchitz test for the following numbers in chain
    while (true)
    {
        devTargetIncrementLength(nProbableChainLength);
	//N = N + N or N *=2
	mpz_set(&N_copy,&N);  
        mpz_mult_u(&N,&N_copy,2);
	// N+ = (fSophieGermain? 1 : (-1))
	mpz_addeq_i(&N,(fSophieGermain? 1 : (-1)));
        if (fFermatTest)
        {
            if (!devFermatProbablePrimalityTest(N, nProbableChainLength))
                break;
        }
        else
        {
            if (!devEulerLagrangeLifchitzPrimalityTest(N, fSophieGermain, nProbableChainLength))
                break;
        }
    }

    mpz_destroy(&N);
    mpz_destroy(&N_copy);

    return (devTargetGetLength(nProbableChainLength) >= 2);
}

// Test probable prime chain for: nOrigin
// Return value:
//   true - Probable prime chain found (one of nChainLength meeting target)
//   false - prime chain too short (none of nChainLength meeting target)
__device__ bool devProbablePrimeChainTest(mpz_t &mpzPrimeChainOrigin, unsigned int nBits, bool fFermatTest, unsigned int& nChainLengthCunningham1, unsigned int& nChainLengthCunningham2, unsigned int& nChainLengthBiTwin)
{
    nChainLengthCunningham1 = 0;
    nChainLengthCunningham2 = 0;
    nChainLengthBiTwin = 0;

    mpz_t mpzPrimeChainOriginMinusOne;
    mpz_t mpzPrimeChainOriginPlusOne;

    mpz_init(&mpzPrimeChainOriginMinusOne);
    mpz_init(&mpzPrimeChainOriginPlusOne);

    mpz_add(&mpzPrimeChainOriginPlusOne,&mpzPrimeChainOrigin,&mpzOne);
    mpz_sub(&mpzPrimeChainOriginMinusOne,&mpzPrimeChainOrigin,&mpzOne);

    // Test for Cunningham Chain of first kind
    devProbableCunninghamChainTest(mpzPrimeChainOriginMinusOne, true, fFermatTest, nChainLengthCunningham1);
    // Test for Cunningham Chain of second kind
    devProbableCunninghamChainTest(mpzPrimeChainOriginPlusOne, false, fFermatTest, nChainLengthCunningham2);
    // Figure out BiTwin Chain length
    // BiTwin Chain allows a single prime at the end for odd length chain
    nChainLengthBiTwin =
        (devTargetGetLength(nChainLengthCunningham1) > devTargetGetLength(nChainLengthCunningham2))?
            (nChainLengthCunningham2 + devTargetFromInt(devTargetGetLength(nChainLengthCunningham2)+1)) :
            (nChainLengthCunningham1 + devTargetFromInt(devTargetGetLength(nChainLengthCunningham1)));

    mpz_destroy(&mpzPrimeChainOriginMinusOne);
    mpz_destroy(&mpzPrimeChainOriginPlusOne);

    return (nChainLengthCunningham1 >= nBits || nChainLengthCunningham2 >= nBits || nChainLengthBiTwin >= nBits);
}

__global__ void runCandidateSearch(cudaCandidateTransfer *candidates, char *result, unsigned int num_candidates)
{
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

	//check bounds
	if (index < num_candidates)
	{
		if(index==0)
			printf("[0] start\n");

		cudaCandidateTransfer *candidate = candidates + index;

		if(index==0)
			printf("[0] candidate is %s\n",candidate->strChainOrigin);

		mpz_t mpzChainOrigin;
		mpz_init(&mpzChainOrigin);
	
		//FIXME: mpz_set_str doesnt work on the device right now
		mpz_set_str(&mpzChainOrigin,candidate->strChainOrigin);

		if(index==0)
			printf("[0] before set_ui\n");

		mpz_init(&mpzOne);
		mpz_set_ui(&mpzOne,1);	

		mpz_init(&mpzTwo);
		mpz_set_ui(&mpzTwo,2);

		mpz_init(&mpzEight);
		mpz_set_ui(&mpzEight,8);	

		if(index==0)
			printf("[0] loaded\n");

		if (devProbablePrimeChainTest(mpzChainOrigin, candidate->blocknBits, false, candidate->nChainLengthCunningham1, candidate->nChainLengthCunningham2, candidate->nChainLengthBiTwin))
		{
			printf("[CUDA] Found probable chain!\n");
			result[index] = 0x01;
		}else
		{
			result[index] = 0x00;
		}

		mpz_destroy(&mpzChainOrigin);
		mpz_destroy(&mpzEight);
		mpz_destroy(&mpzOne);
	}

}

void runCandidateSearchKernel(cudaCandidateTransfer *candidates, char *result, unsigned int num_candidates)
{
	//TODO: make gridsize dynamic
	runCandidateSearch<<< 200 , 50>>>(candidates, result, num_candidates);
}
